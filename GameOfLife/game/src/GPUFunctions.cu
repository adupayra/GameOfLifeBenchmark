#include "hip/hip_runtime.h"
#include "GPUFunctions.cuh"
#include <hip/hip_runtime.h>
#include ""
#include <algorithm>
#include <iostream>

void gpuAssert(hipError_t code)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code));
		exit(EXIT_FAILURE);
	}
}

__global__ void processCellKernel(uint8_t* cells, uint8_t* newCells, int dimensions, int cellsPerDim, int nbCells) {
	int cellID = threadIdx.x + blockIdx.x * blockDim.x;

	int nbNeighbours = 0;

	int colNb = cellID % cellsPerDim;
	int rowNb = cellID / cellsPerDim;
	int newCell = cellID + 1;

	int x0 = (rowNb - 1 + cellsPerDim) % cellsPerDim;
	int x2 = (rowNb + 1) % cellsPerDim;
	int y0 = (colNb - 1 + cellsPerDim) % cellsPerDim;
	int y2 = (colNb + 1) % cellsPerDim;

	nbNeighbours += cells[x0 * cellsPerDim + y0];
	nbNeighbours += cells[x0 * cellsPerDim + colNb];
	nbNeighbours += cells[x0 * cellsPerDim + y2];
	nbNeighbours += cells[rowNb * cellsPerDim + y0];
	nbNeighbours += cells[rowNb * cellsPerDim + y2];
	nbNeighbours += cells[x2 * cellsPerDim + y0];
	nbNeighbours += cells[x2 * cellsPerDim + colNb];
	nbNeighbours += cells[x2 * cellsPerDim + y2];
	
	bool newValue = false;
	if (cells[cellID]) {
		newValue = nbNeighbours == 2 || nbNeighbours == 3;
	} else
		newValue = nbNeighbours == 3;

	newCells[cellID] = newValue;
}


void processCells(uint8_t*& cells, uint8_t*& devCells, uint8_t*& devNewCells, int dimensions, int cellsPerDim, int nbCells, int nbBlocks, int nbThreads) {

	processCellKernel << <nbBlocks, nbThreads >> > (devCells, devNewCells, dimensions, cellsPerDim, nbCells);
	gpuAssert(hipPeekAtLastError());
	gpuAssert(hipDeviceSynchronize());
	gpuAssert(hipMemcpy(cells, devNewCells, nbCells, hipMemcpyDeviceToHost));
	std::swap(devCells, devNewCells);

}